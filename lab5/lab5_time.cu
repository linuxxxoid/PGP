#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <chrono>


#define BLOCK_COUNT 256u
#define HALF_BLOCK_COUNT 128u
#define BANKS 16
#define LOG_2_BANKS 4
// macro used for computing
// Bank-Conflict-Free Shared Memory Array Indices
#define AVOID_BANK_CONFLICTS(idx) ((idx) >> BANKS + (idx) >> (LOG_2_BANKS << 1))

#define CSC(call) do { \
	hipError_t res = call;	\
	if (res != hipSuccess) { \
		fprintf(stderr, "CUDA Error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(res)); \
		exit(0); \
	} \
} while (0)


__global__ void Histogram(unsigned char* data, int size, int* histo)
{
	// выделяем разделяемую память, объем памяти равен количеству корзинок
	__shared__ int tmp[BLOCK_COUNT];

	// вычисляем абсолютный идентификатор 
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	// смещение
	int offset = gridDim.x * blockDim.x;

	// заполним временный массив нулем,
	// фактически заполняем 256 элементов как 0 в общей памяти
	tmp[threadIdx.x] = 0;
	__syncthreads(); // потоки ожидают выполнения заполнения нулем tmp
	
	// перебираем все элементы буфера data,
	// пока абс идентификатор не коснется значения size
	while (idx < size)
	{
		// извлекаем значение, находящееся в буфере
		// и увеличиваем счетчик в массиве разделяемой памяти
		atomicAdd(&tmp[data[idx]], 1);
		idx += offset; // увеличение абс идентиф по смещение
	}
	__syncthreads(); // ждем все потоки

	// обновляем окончательный результат в массиве histo
	int i = threadIdx.x;
	while (i < BLOCK_COUNT)
	{
		atomicAdd(&histo[i], tmp[i]);
		i += blockDim.x;
	}
}


__global__ void Scan(int* histo, int* prefixSum)
{
	__shared__ int tmp[BLOCK_COUNT];

	int threadId = threadIdx.x;
	int offset = 1;

	int aIdx = threadIdx.x;
	int bIdx = threadIdx.x + HALF_BLOCK_COUNT;

	int bankOffsetA = AVOID_BANK_CONFLICTS(aIdx);
	int bankOffsetB = AVOID_BANK_CONFLICTS(bIdx);

	// загружаем данные из гистограммы в общую память
	tmp[aIdx + bankOffsetA] = histo[aIdx];
	tmp[bIdx + bankOffsetB] = histo[bIdx];

	// строим сумму на месте вверх по дереву
	{
		int lvl = BLOCK_COUNT >> 1;

		while (lvl > 0)
		{
			__syncthreads();

			if (threadId < lvl)
			{
				int aIndex = (offset * (threadId * 2 + 1) - 1);
				int bIndex = (offset * (threadId * 2 + 2) - 1);
				aIndex += AVOID_BANK_CONFLICTS(aIndex);
				bIndex += AVOID_BANK_CONFLICTS(bIndex); 
				tmp[bIndex] += tmp[aIndex];
			}
			offset <<= 1;
			lvl >>= 1;
		}
	}

	// очищаем последний элемент
	if (threadId == 0)
	{
		tmp[BLOCK_COUNT - 1 + AVOID_BANK_CONFLICTS(BLOCK_COUNT - 1)] = 0;
	}

	// идем вниз по "дереву" и строим сканирование
	{
		int lvl = 1; 
		while (lvl < BLOCK_COUNT)
		{
			offset >>= 1;
			__syncthreads();
			if (threadId < lvl)
			{
				int aIndex = (offset * (threadId * 2 + 1) - 1);
				int bIndex = (offset * (threadId * 2 + 2) - 1);
				aIndex += AVOID_BANK_CONFLICTS(aIndex);
				bIndex += AVOID_BANK_CONFLICTS(bIndex);
				int temp = tmp[aIndex];
				tmp[aIndex] = tmp[bIndex];
				tmp[bIndex] += temp; 
			}
			lvl <<= 1;
		}
	} 

	__syncthreads();
	// записываем результаты в массив prefixSum
	prefixSum[aIdx] = histo[aIdx] + tmp[aIdx + bankOffsetA];
	prefixSum[bIdx] = histo[bIdx] + tmp[bIdx + bankOffsetB];
	
}



__global__ void CountSort(unsigned char* data, int* prefixSum, unsigned char* result, int size)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = gridDim.x * blockDim.x;

	int i = idx, j;
	while (i < size)
	{
		// j = prefixSum[i] - 1;
		// bound = i ? prefixSum[i - 1] : 0;

		// while (j >= bound)
		// {
		// 	data[j] = i;
		// 	--j;
		// }
		j = atomicSub(&prefixSum[data[i]], 1) - 1;
		result[j] = data[i];
		i += offset;
	}
}


int main()
{
	int size;

	freopen(NULL, "rb", stdin);
	fread(&size, sizeof(int), 1, stdin);

    unsigned char* data = new unsigned char[size];
	
	fread(data, sizeof(unsigned char), size, stdin);
	fclose(stdin);

    unsigned char* deviceData;
    unsigned char* deviceResult;
    int* deviceHisto;
    int* devicePrefix;

	float elapsedTime;
	hipEvent_t start, stop;

	CSC(hipEventCreate(&start));
	CSC(hipEventCreate(&stop));

	CSC(hipMalloc((void**)&deviceData, sizeof(unsigned char) * size));
	CSC(hipMemcpy(deviceData, data, sizeof(unsigned char) * size, hipMemcpyHostToDevice));

	CSC(hipMalloc((void**)&deviceHisto, sizeof(int) * BLOCK_COUNT));
	CSC(hipMalloc((void**)&devicePrefix, sizeof(int) * BLOCK_COUNT));
	CSC(hipMemset(deviceHisto, 0, sizeof(int) * BLOCK_COUNT));

	CSC(hipMalloc((void**)&deviceResult, sizeof(unsigned char) * size));

	CSC(hipEventRecord(start));

	Histogram<<<BLOCK_COUNT, BLOCK_COUNT>>>(deviceData, size, deviceHisto);
    hipDeviceSynchronize(); // wait end
	CSC(hipGetLastError());

	Scan<<<1, HALF_BLOCK_COUNT>>>(deviceHisto, devicePrefix);
    hipDeviceSynchronize(); // wait end
	CSC(hipGetLastError());

	CountSort<<<1, BLOCK_COUNT>>>(deviceData, devicePrefix, deviceResult, size);
    hipDeviceSynchronize(); // wait end
	CSC(hipGetLastError());

	CSC(hipEventRecord(stop));
	CSC(hipEventSynchronize(stop));

	CSC(hipEventElapsedTime(&elapsedTime, start, stop));
	CSC(hipEventDestroy(start));
	CSC(hipEventDestroy(stop));
	printf("Time :  %f ms \n", elapsedTime);

	CSC(hipMemcpy(data, deviceResult, sizeof(unsigned char) * size, hipMemcpyDeviceToHost));

	// freopen(NULL, "wb", stdout);
	// fwrite(data, sizeof(unsigned char), size, stdout);
	// fclose(stdout);

	CSC(hipFree(deviceData));
	CSC(hipFree(deviceHisto));
	CSC(hipFree(devicePrefix));

	delete[] data;
	return 0;
}