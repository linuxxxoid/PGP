
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <stdlib.h>


void checkCudaError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "ERROR: %s: %s.\n", msg, hipGetErrorString(err));
        exit(0);
    }
}


__global__ void Reverse(float* res, float* vec, int size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= size) return;
    int offset = size - idx - 1;
    res[idx] = vec[offset];
}


int main(int argc, const char* argv[])
{
    int size;
    std::cin >> size;
    
    const int MAX = 33554432;
    const int MIN = 0;
    if (size < MIN && size > MAX)
    {
       std::cerr << "ERROR: Incorrect size!\n";
       exit(0);
    }

    float *hostVec = new float[size];

    for (int i = 0; i < size; ++i)
    {
        hostVec[i] = i;
    }

    float *deviceVec, *deviceRes;

    // Выделяем память для device копий
    hipMalloc((void**) &deviceVec, sizeof(float) * size);
    hipMalloc((void**) &deviceRes, sizeof(float) * size);
    // Копируем ввод на device
    hipMemcpy(deviceVec, hostVec, sizeof(float) * size, hipMemcpyHostToDevice);
    
    const int maxThreads = 1024;
    int blockCount = size / maxThreads;
    int threadsCount;
    
    if (blockCount * maxThreads != size) 
        ++blockCount; 

    if (size < maxThreads)
        threadsCount = size;
    else
        threadsCount = maxThreads;    
    

    hipEvent_t start, end;
    hipEventCreate(&start);
    checkCudaError("hipEventCreate");
    hipEventCreate(&end);
    checkCudaError("hipEventCreate");
    hipEventRecord(start);
    checkCudaError("hipEventRecord");


    // Запускаем kernel
    Reverse<<<blockCount, threadsCount>>>(deviceRes, deviceVec, size);
    checkCudaError("Kernel invocation");
    
    hipEventRecord(end);
    checkCudaError("hipEventRecord");
    hipEventSynchronize(end);
    checkCudaError("hipEventSynchronize");
    float t;
    hipEventElapsedTime(&t, start, end);
    checkCudaError("hipEventElapsedTime");
    hipEventDestroy(start);
    checkCudaError("hipEventDestroy");
    hipEventDestroy(end);
    checkCudaError("hipEventDestroy");
    printf("time = %f\n", t);

    hipMemcpy(hostVec, deviceRes, sizeof(float) * size, hipMemcpyDeviceToHost);
    checkCudaError("Memcpy");

    const int accuracy = 10;
    for (int i = 0; i < size - 1; ++i)
    {
        //std::cout << std::scientific << std::setprecision(accuracy) << hostVec[i] << " ";
    }
    std::cout << std::scientific << std::setprecision(accuracy) << hostVec[size - 1];
    
    hipFree(deviceVec);
    checkCudaError("Free");
    
    hipFree(deviceRes);
    checkCudaError("Free");
    
    delete[] hostVec;

    return 0;
}