
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <stdlib.h>


void checkCudaError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "ERROR: %s: %s.\n", msg, hipGetErrorString(err));
        exit(0);
    }
}


__global__ void Reverse(float* res, float* vec, int size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= size) return;
    int offset = size - idx - 1;
    res[idx] = vec[offset];
}


int main(int argc, const char* argv[])
{
    int size;
    std::cin >> size;
    
    const int MAX = 33554432;
    const int MIN = 0;
    if (size < MIN && size > MAX)
    {
       std::cerr << "ERROR: Incorrect size!\n";
       exit(0);
    }

    float *hostVec = new float[size];

    for (int i = 0; i < size; ++i)
    {
        std::cin >> hostVec[i];
    }

    float *deviceVec, *deviceRes;

    // Выделяем память для device копий
    hipMalloc((void**) &deviceVec, sizeof(float) * size);
    hipMalloc((void**) &deviceRes, sizeof(float) * size);
    // Копируем ввод на device
    hipMemcpy(deviceVec, hostVec, sizeof(float) * size, hipMemcpyHostToDevice);
    
    const int maxThreads = 1024;
    int blockCount = size / maxThreads;
    int threadsCount;
    
    if (blockCount * maxThreads != size) 
        ++blockCount; 

    if (size < maxThreads)
        threadsCount = size;
    else
        threadsCount = maxThreads;    
    
    // Запускаем kernel
    Reverse<<<blockCount, threadsCount>>>(deviceRes, deviceVec, size);
    checkCudaError("Kernel invocation");
    
    hipMemcpy(hostVec, deviceRes, sizeof(float) * size, hipMemcpyDeviceToHost);
    checkCudaError("Memcpy");

    const int accuracy = 10;
    for (int i = 0; i < size; ++i)
    {
        std::cout << std::scientific << std::setprecision(accuracy) << hostVec[i];
        if (i < size - 1)
            std::cout << " ";
        else
            std::cout << std::endl;
    }
    hipFree(deviceVec);
    checkCudaError("Free");
    
    hipFree(deviceRes);
    checkCudaError("Free");
    
    delete[] hostVec;

    return 0;
}
