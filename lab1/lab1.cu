
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <stdlib.h>


void checkCudaError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "ERROR: %s: %s.\n", msg, hipGetErrorString(err));
        exit(0);
    }
}


__global__ void Reverse(float* res, float* vec, int size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;

    while (idx < size)
    {
	    res[idx] = vec[size - idx - 1];
	    idx += offset;
    }
}


int main(int argc, const char* argv[])
{
    int size;
    std::cin >> size;
    
    const int MAX = 33554432;
    const int MIN = 0;
    if (size < MIN && size > MAX)
    {
       std::cerr << "ERROR: Incorrect size!\n";
       exit(0);
    }

    float *hostVec = new float[size];

    for (int i = 0; i < size; ++i)
    {
        std::cin >> hostVec[i];
    }

    float *deviceVec, *deviceRes;

    // Выделяем память для device копий
    hipMalloc((void**) &deviceVec, sizeof(float) * size);
    checkCudaError("Malloc");

    hipMalloc((void**) &deviceRes, sizeof(float) * size);
    checkCudaError("Malloc");

    // Копируем ввод на device
    hipMemcpy(deviceVec, hostVec, sizeof(float) * size, hipMemcpyHostToDevice);
    checkCudaError("Memcpy");
    
    int blockCount = 256;
    int threadsCount = 256;   
    
    // Запускаем kernel
    Reverse<<<blockCount, threadsCount>>>(deviceRes, deviceVec, size);
    checkCudaError("Kernel invocation");
    
    hipMemcpy(hostVec, deviceRes, sizeof(float) * size, hipMemcpyDeviceToHost);
    checkCudaError("Memcpy");

    const int accuracy = 10;
    for (int i = 0; i < size - 1; ++i)
    {
        std::cout << std::scientific << std::setprecision(accuracy) << hostVec[i] << " ";
    }
    std::cout << std::scientific << std::setprecision(accuracy) << hostVec[size - 1];
    
    hipFree(deviceVec);
    checkCudaError("Free");
    
    hipFree(deviceRes);
    checkCudaError("Free");
    
    delete[] hostVec;

    return 0;
}