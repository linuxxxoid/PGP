
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <stdlib.h>


void checkCudaError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "ERROR: %s: %s.\n", msg, hipGetErrorString(err));
        exit(0);
    }
}


__global__ void Reverse(double* res, double* vec, int size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= size) return;
    int offset = gridDim.x * blockDim.x - 1 - idx;
    res[idx] = vec[offset];
}


int main(int argc, const char* argv[])
{
    int size;
    std::cin >> size;
    
    const int MAX = 33554432;
    const int MIN = 0;
    if (size < MIN && size > MAX)
    {
       std::cerr << "ERROR: Incorrect size!\n";
       exit(0);
    }

    double *hostVec = new double[size];

    for (int i = 0; i < size; ++i)
    {
        std::cin >> hostVec[i];
    }

    double *deviceVec, *deviceRes;

    // Выделяем память для device копий
    hipMalloc((void**) &deviceVec, sizeof(double) * size);
    hipMalloc((void**) &deviceRes, sizeof(double) * size);
    // Копируем ввод на device
    hipMemcpy(deviceVec, hostVec, sizeof(double) * size, hipMemcpyHostToDevice);
    
    const int maxThreads = 1024;
    int blockCount = size / maxThreads;
    
    if (blockCount * maxThreads != size) 
    {
        ++blockCount; 
    }
   
    // Запускаем kernel
    Reverse<<<blockCount, maxThreads>>>(deviceRes, deviceVec, size);

    checkCudaError("Kernel invocation");
    hipMemcpy(hostVec, deviceRes, sizeof(double) * size, hipMemcpyDeviceToHost);
    checkCudaError("Memcpy");

    const int accuracy = 10;
    for (int i = 0; i < size; ++i)
    {
        std::cout << std::scientific << std::setprecision(accuracy) << hostVec[i];
        if (i < size - 1)
            std::cout << " ";
        else
            std::cout << std::endl;
    }
    hipFree(deviceVec);
    hipFree(deviceRes);
    delete[] hostVec;

    return 0;
}
