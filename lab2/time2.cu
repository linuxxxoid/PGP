#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
/*

void CustomSSAA(int* oldPixels, int width, int height, int* colorPixels, int newidth, int height, int proportionWidth, int proportionHeight)
{
	int numSample = proportionWidth * proportionHeight;
	int colorSample = 0;

	for (int x = 0; x < width; x += proportionWidth) 
	{
		for (int y = 0; y < height; y += proportionHeight)
		{
			colorSample = 0;
			for (int i = 0; i < proportionWidth; ++i)
			{
				for (int j = 0; j < proportionHeight; ++j)
				{
					colorSample += oldPixels[x * proportionWidth + i] + oldPixels[y *  proportionHeight + j];
				}
				
			}
			colorSample /= numSample;
			colorPixels[x + y * width] = colorSample;
		}
	}
}
*/


texture<uchar4, 2, hipReadModeElementType> Texture2D;


void checkCudaError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "ERROR: %s: %s!\n", msg, hipGetErrorString(err));
        exit(0);
    }
}



__global__ void SSAA(uchar4 *colorPixels, int width, int height, int proportionWidth, int proportionHeight)
{
	int xId = blockDim.x * blockIdx.x + threadIdx.x;
	int yId = blockDim.y * blockIdx.y + threadIdx.y;
	int xOffset = blockDim.x * gridDim.x;
	int yOffset = blockDim.y * gridDim.y;
	int numSample = proportionWidth * proportionHeight;

	/*
	colorPixels = colorSample_0 + colorSample_1 + ... + colorSample_n-1   SUM(colorSample_i)
		          ----------------------------------------------------- = ------------------
	                            numSample                                   numSample
	 
	 colorPixels - the final color of the pixel,
	 numSample - the number of samples per pixel,
	 colorSample_i - color of the i-th sample.
	*/

	for (int col = xId; col < width; col += xOffset)
	{
		for (int row = yId; row < height; row += yOffset)
		{
			int3 colorSample;
			colorSample.x = 0;
			colorSample.y = 0;
			colorSample.z = 0;
			for (int i = 0; i < proportionWidth; ++i)
			{
				for (int j = 0; j < proportionHeight; ++j)
				{
					uchar4 pix = tex2D(Texture2D, col * proportionWidth + i, row * proportionHeight + j);
					colorSample.x += pix.x;
					colorSample.y += pix.y;
					colorSample.z += pix.z;
				}
			}
			colorSample.x /= numSample; 
			colorSample.y /= numSample;
			colorSample.z /= numSample;
            // Write to global memory
			colorPixels[col + row * width] = make_uchar4(colorSample.x, colorSample.y, colorSample.z, 0);
		}
	}
}



int main(int argc, const char* argv[])
{	
	std::string input, output;
	int widthNew, heightNew, width, height;
	uchar4 *pixels;
	std::cin >> input >> output >> widthNew >> heightNew;

	FILE* file;
	if ((file = fopen(input.c_str(), "rb")) == NULL)
	{
	    std::cerr << "ERROR: something wrong with opening the file!\n";
        exit(0);
	}
	else
	{
		fread(&width, sizeof(int), 1, file);
		fread(&height, sizeof(int), 1, file);
		if (width >= 65536 || width < 0 || height < 0 || height >= 65536)
		{
			std::cerr << "ERROR: incorrect input.\n";
			exit(0);
		}
		pixels = new uchar4[width * height];
		fread(pixels, sizeof(uchar4), width * height, file);

		fclose(file);
	}

	int proportionWidth = width / widthNew;
	int proportionHeight = height / heightNew;


	int cpu_width, cpu_height;
	int* cpu_pixels;
	if ((file = fopen(input.c_str(), "rb")) == NULL)
	{
	    std::cerr << "ERROR: something wrong with opening the file!\n";
        exit(0);
	}
	else
	{
		fread(&cpu_width, sizeof(int), 1, file);
		fread(&cpu_height, sizeof(int), 1, file);
		if (cpu_width >= 65536 || cpu_width < 0 || cpu_height < 0 || cpu_height >= 65536)
		{
			std::cerr << "ERROR: incorrect input.\n";
			exit(0);
		}
		cpu_pixels = new int[cpu_width * cpu_height];
		fread(cpu_pixels, sizeof(int), cpu_width * cpu_height, file);

		fclose(file);
	}
	clock_t time;
	time = clock();

	int* newPixels = new int[widthNew * heightNew];
	//CustomSSAA(cpu_pixels, width, height, newPixels, widthNew, heightNew, proportionWidth, proportionHeight);
	time = clock() - time;
	std::cout << "CPU" << std::endl;
	std::cout << "time = " << (double)time/CLOCKS_PER_SEC << std::endl;


    // Allocate CUDA array in device memory
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar4>();
	hipArray *array;

	hipMallocArray(&array, &channelDesc, width, height);
	checkCudaError("Malloc array");

	hipMemcpyToArray(array, 0, 0, pixels, sizeof(uchar4) * width * height, hipMemcpyHostToDevice);
	checkCudaError("Memcpy array");

	// set texture parameters
	Texture2D.addressMode[0] = hipAddressModeClamp;
	Texture2D.addressMode[1] = hipAddressModeClamp;
	Texture2D.channelDesc = channelDesc;
	Texture2D.filterMode = hipFilterModePoint;;
	Texture2D.normalized = false; // access with normalized texture coordinates

	// Bind the array to the texture
	hipBindTextureToArray(Texture2D, array, channelDesc);
	checkCudaError("Bind");

	uchar4 *deviceRes;
	hipMalloc(&deviceRes, sizeof(uchar4) * widthNew * heightNew);
	checkCudaError("Malloc");


    int xThreadCount = 8;
    int yThreadCount = 8;

	int xBlockCount = 8;
	int yBlockCount = 8;

 
    dim3 blockCount = dim3(xBlockCount, yBlockCount);
    dim3 threadsCount = dim3(xThreadCount, yThreadCount);


	hipEvent_t start, end;
    hipEventCreate(&start);
    checkCudaError("hipEventCreate");
    hipEventCreate(&end);
    checkCudaError("hipEventCreate");
    hipEventRecord(start);
    checkCudaError("hipEventRecord");


	SSAA<<<blockCount, threadsCount>>>(deviceRes, widthNew, heightNew, proportionWidth, proportionHeight);
	checkCudaError("Kernel invocation");


	hipEventRecord(end);
    checkCudaError("hipEventRecord");
    hipEventSynchronize(end);
    checkCudaError("hipEventSynchronize");
    float t;
    hipEventElapsedTime(&t, start, end);
    checkCudaError("hipEventElapsedTime");
    hipEventDestroy(start);
    checkCudaError("hipEventDestroy");
    hipEventDestroy(end);
    checkCudaError("hipEventDestroy");
    printf("GPU\n");
    printf("time = %f\n", t);
    printf("blocks = %d\n", xBlockCount * yBlockCount);
    printf("threads = %d\n", xThreadCount * yThreadCount);


	hipMemcpy(pixels, deviceRes, sizeof(uchar4) * widthNew * heightNew, hipMemcpyDeviceToHost);
	checkCudaError("Memcpy");
	

	if ((file = fopen(output.c_str(), "wb")) == NULL)
	{
		std::cerr << "ERROR: something wrong with opening the file.";
        exit(0);
	}
	else
	{
		fwrite(&widthNew, sizeof(int), 1, file);
		fwrite(&heightNew, sizeof(int), 1, file);
		fwrite(pixels, sizeof(uchar4), widthNew * heightNew, file);
		fclose(file);
	}

	std::string cpu_output = "cpu_a.data";
	if ((file = fopen(cpu_output.c_str(), "wb")) == NULL)
	{
		std::cerr << "ERROR: something wrong with opening the file.";
        exit(0);
	}
	else
	{
		fwrite(&widthNew, sizeof(int), 1, file);
		fwrite(&heightNew, sizeof(int), 1, file);
		fwrite(newPixels, sizeof(int), widthNew * heightNew, file);
		fclose(file);
	}

	hipUnbindTexture(Texture2D);
	checkCudaError("Unbind");

	hipFreeArray(array);
	checkCudaError("Free");
	
	hipFree(deviceRes);
	checkCudaError("Free");

	delete[] pixels;
	delete[] cpu_pixels;
	delete[] newPixels;
	return 0;
}